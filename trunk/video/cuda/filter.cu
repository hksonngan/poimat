#include "hip/hip_runtime.h"
#include "filter.h"
#include "kernel.h"
#include "alpha.h"
#include "flood.h"

#define TEX_NUM 8
hipArray* cuArray[TEX_NUM];

//
//  interface
//

static bool initialized = false;
static float *alpha = NULL;
static float2 *nearest = NULL;
static float4 *image = NULL;

extern "C" void
initializeTexture(
    unsigned int w,
    unsigned int h
) {
    if (initialized) {
        releaseTexture();
    }

    hipChannelFormatDesc channelDescB4 = hipCreateChannelDesc<uchar4>();
    hipChannelFormatDesc channelDescB = hipCreateChannelDesc<unsigned char>();
    hipChannelFormatDesc channelDescF4 = hipCreateChannelDesc<float4>();
    hipChannelFormatDesc channelDescF2 = hipCreateChannelDesc<float2>();
    hipChannelFormatDesc channelDescF = hipCreateChannelDesc<float>();

    hipMallocArray(&cuArray[0], &channelDescB4, w, h); // image
    hipMallocArray(&cuArray[1], &channelDescB,  w, h); // trimap
    hipMallocArray(&cuArray[2], &channelDescF4, w, h); // temp image
    hipMallocArray(&cuArray[3], &channelDescF4, w, h); // foreground
    hipMallocArray(&cuArray[4], &channelDescF4, w, h); // background
    hipMallocArray(&cuArray[5], &channelDescF2, w, h); // nearest pixel
    hipMallocArray(&cuArray[6], &channelDescF,  w, h); // laplacian alpha
    hipMallocArray(&cuArray[7], &channelDescF,  w, h); // alpha

    bindFloodTexture(cuArray[0], cuArray[7], cuArray[5]);
    bindAlphaTexture(cuArray[0], cuArray[1], cuArray[3], cuArray[4], cuArray[6], cuArray[7]);

    hipMalloc((void**)&alpha,   w*h*sizeof(float));
    hipMalloc((void**)&nearest, w*h*sizeof(float2));
    hipMalloc((void**)&image,   w*h*sizeof(float4));

    initialized = true;
}

extern "C" void
releaseTexture(
) {
    if (!initialized) {
        return;
    }

    unbindFloodTexture();
    unbindAlphaTexture();

    for (int i=0; i<TEX_NUM; i++){
        hipFreeArray(cuArray[i]);
    }

    hipFree(alpha);
    hipFree(nearest);
    hipFree(image);

    initialized = false;
}

extern "C" void
poissonFilter(
    const unsigned char *h_src,
    const unsigned char *h_trimap,
    float *d_dst,
    size_t w,
    size_t h
) {
    dim3 grid(div(w,BLOCK_SIZE), div(h,BLOCK_LEN));
    int threads = min(BLOCK_SIZE, w);
    int maxJump;
    for (maxJump=1; maxJump<w || maxJump<h; maxJump*=2);

    // cuda array 0: image
    hipMemcpyToArray(cuArray[0], 0, 0, h_src, w*h*sizeof(uchar4), hipMemcpyHostToDevice);

    // cuda array 1: trimap
    hipMemcpyToArray(cuArray[1], 0, 0, h_trimap, w*h*sizeof(unsigned char), hipMemcpyHostToDevice);

    // cuda array 7: alpha
    alphaFromTrimap <<< grid, threads, 0 >>> (alpha, w, h);
    hipMemcpyToArray(cuArray[7], 0, 0, alpha, w*h*sizeof(float), hipMemcpyDeviceToDevice);

    for (int time=0; time<4; time++) {

    /*
     *  Calculate Foreground
     */

    preflood  <<< grid, threads, 0 >>> (image, nearest, 1.0f, w, h);
    for (int step=maxJump; step>0; step/=2) {
        // cuda array 5: nearest pixel
        hipMemcpyToArray(cuArray[5], 0, 0, nearest, w*h*sizeof(float2), hipMemcpyDeviceToDevice);
        flood <<< grid, threads, 0 >>> (nearest, step, w, h);
    }
    // cuda array 5: nearest pixel
    hipMemcpyToArray(cuArray[5], 0, 0, nearest, w*h*sizeof(float2), hipMemcpyDeviceToDevice);
    postflood <<< grid, threads, 0 >>> (image, w, h);
    // cuda array 3: forground image
    hipMemcpyToArray(cuArray[3], 0, 0, image, w*h*sizeof(float4), hipMemcpyDeviceToDevice);

    /*
     *  Calculate Background
     */

    preflood  <<< grid, threads, 0 >>> (image, nearest, 0.0f, w, h);
    for (int step=maxJump; step>0; step/=2) {
        // cuda array 5: nearest pixel
        hipMemcpyToArray(cuArray[5], 0, 0, nearest, w*h*sizeof(float2), hipMemcpyDeviceToDevice);
        flood <<< grid, threads, 0 >>> (nearest, step, w, h);
    }
    // cuda array 5: nearest pixel
    hipMemcpyToArray(cuArray[5], 0, 0, nearest, w*h*sizeof(float2), hipMemcpyDeviceToDevice);
    postflood <<< grid, threads, 0 >>> (image, w, h);
    // cuda array 4: background image
    hipMemcpyToArray(cuArray[4], 0, 0, image, w*h*sizeof(float4), hipMemcpyDeviceToDevice);

    /*
     *  Calculate Alpha
     */

    // texture 6: laplacian alpha
    alphaGradient <<< grid, threads, 0 >>> (alpha, w, h);
    hipMemcpyToArray(cuArray[6], 0, 0, alpha, w*h*sizeof(float), hipMemcpyDeviceToDevice);
    alphaInitialize <<< grid, threads, 0 >>> (alpha, w, h);
    for (int i=0; i<64; i++){
        // texture 7: alpha
        hipMemcpyToArray(cuArray[7], 0, 0, alpha, w*h*sizeof(float), hipMemcpyDeviceToDevice);
        alphaReconstruct <<< grid, threads, 0 >>> (alpha, w, h);
    }
    alphaRefinement <<< grid, threads, 0 >>> (alpha, w, h);
    // texture 7: alpha
    hipMemcpyToArray(cuArray[7], 0, 0, alpha, w*h*sizeof(float), hipMemcpyDeviceToDevice);

    }

    // output alpha
    alphaOutput <<< grid, threads, 0 >>> ((float4*)d_dst, w, h);
}
